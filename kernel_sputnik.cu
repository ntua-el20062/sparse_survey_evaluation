#include <stdio.h>
#include <stdlib.h>

#include "macros/cpp_defines.h"

#include "bench_common.h"
#include "kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

#include <hip/hip_runtime.h>

#ifdef SPMM_KERNEL
	#include "sputnik/spmm/cuda_spmm.h"   // CudaSpmm
	using namespace sputnik;
#endif
#ifdef SDDMM_KERNEL
	#include "sputnik/sddmm/cuda_sddmm.h"   // CudaSddmm
	using namespace sputnik;
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;

	hipStream_t stream;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * out = NULL;

	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	ValueType * out_d = NULL;

	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	ValueType * out_h = NULL;

	int *row_idx, *row_idx_d;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipMalloc((void**)&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&a_d, nnz * sizeof(*a_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));

		gpuCudaErrorCheck(hipHostMalloc((void**)&ia_h, (m+1) * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&ja_h, nnz * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&a_h, nnz * sizeof(*a_h)));

		memcpy(ia_h, ia, (m+1) * sizeof(*ia_h));
		memcpy(ja_h, ja, nnz * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));

		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));

		// wait for transfers to finish
		gpuCudaErrorCheck(hipStreamSynchronize(stream));

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		// compute row_idx_d (just the starting position of each row, before being reordered in the function called)
		row_idx = (typeof(row_idx)) malloc(m * sizeof(*row_idx));
		for(int i=0; i<m; i++)
			row_idx[i] = i;
		gpuCudaErrorCheck(hipMalloc((void**) &row_idx_d, m * sizeof(int)));
		gpuCudaErrorCheck(hipMemcpy(row_idx_d, row_idx, m * sizeof(int), hipMemcpyHostToDevice));

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(out_d));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));
		gpuCudaErrorCheck(hipHostFree(out_h));

		gpuCudaErrorCheck(hipStreamDestroy(stream));

		gpuCudaErrorCheck(hipFree(row_idx_d));
		free(row_idx);
	}

	void spmm(ValueType * x, ValueType * y, int k);
	void sddmm(ValueType * x, ValueType * y, ValueType * out, int k);
};

void compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k);
void compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k);

void
CSRArrays::spmm(ValueType * x, ValueType * y, int k)
{
	compute_spmm(this, x, y, k);
}

void
CSRArrays::sddmm(ValueType * x, ValueType * y, ValueType * out, int k)
{
	compute_sddmm(this, x, y, out, k);
}

struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "Sputnik";
	return csr;
}

//==========================================================================================================================================
//= Computation
//==========================================================================================================================================

void
compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k)
{
	__attribute__((unused)) const ValueType alpha = 1.0;
	__attribute__((unused)) const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->n * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->n * k * sizeof(*csr->x_h)));

		memcpy(csr->x_h, x, csr->n * k * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Also, prepare for the output matrix y
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, csr->m * k * sizeof(*csr->y_d)));
	}

	#ifdef SPMM_KERNEL
	CudaSpmm(csr->m, csr->n, k, csr->nnz, csr->row_idx_d, csr->a_d, csr->ia_d, csr->ja_d, csr->x_d, csr->y_d, 0);
	#endif

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, csr->m * k * sizeof(*csr->y_h)));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * k * sizeof(*csr->y_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * k * sizeof(ValueType));
	}
}

void
compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k)
{
	__attribute__((unused)) const ValueType alpha = 1.0;
	__attribute__((unused)) const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		csr->y = y;

		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->m * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, k * csr->n * sizeof(*csr->y_d)));

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->m * k * sizeof(*csr->x_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, k * csr->n * sizeof(*csr->y_h)));

		memcpy(csr->x_h, x, csr->m * k * sizeof(ValueType));
		memcpy(csr->y_h, y, k * csr->n * sizeof(ValueType));

		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->m * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_d, csr->y_h, k * csr->n * sizeof(*csr->y_d), hipMemcpyHostToDevice, csr->stream));

		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Also, prepare for the output values
		// gpuCudaErrorCheck(hipMalloc((void**)&csr->out_d, csr->nnz * sizeof(*csr->out_d)));
	}

	#ifdef SDDMM_KERNEL
	CudaSddmm(csr->m, k, csr->n, csr->nnz, csr->row_idx_d, csr->ia_d, csr->ja_d, csr->x_d, csr->y_d, csr->a_d, 0);
	#endif

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->out == NULL)
	{
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->out_h, csr->nnz * sizeof(*csr->out_h)));

		csr->out = out;

		gpuCudaErrorCheck(hipMemcpyAsync(csr->out_h, csr->a_d, csr->nnz * sizeof(*csr->a_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(out, csr->out_h, csr->nnz * sizeof(ValueType));
	}
}
