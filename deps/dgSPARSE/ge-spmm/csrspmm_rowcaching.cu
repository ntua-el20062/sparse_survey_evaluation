#include "hip/hip_runtime.h"
// file: csrspmm_rowcaching.cuh
//      Implementation of row-caching kernels

#include "../util/cuda_util.cuh"
#include "gespmm.h"

// Row-caching strategy pre-loads sparse elements into shared memory
// bucket-by-bucket and share the buffered sparse values within the same warp.
// The __syncwarp() primitive is used to assure shared-memory race safety.

template <int CoarsenFactor>
__global__ void csrspmm_rowcaching_rowbalance_kernel(
    const int M, const int N, const int K, const int csr_indptr[],
    const int csr_indices[], const float csr_data[], const float B[],
    float C[]) {
  int warp_id = threadIdx.x >> 5;
  int lane_id = threadIdx.x & 31;

  extern __shared__ int shared_mem[];
  int *workspace_indices = &shared_mem[(warp_id << 5)];
  float *workspace_data =
      (float *)(workspace_indices +
                blockDim.x); // float and int has the same size

  // get the sparse-value range of this row
  int row_id = blockIdx.x * (blockDim.x >> 5) + warp_id;
  if (row_id >= M)
    return;
  int start = csr_indptr[row_id];
  int end = csr_indptr[row_id + 1];

  // get the dense column offset
  int col_offset = blockIdx.y * 32 * CoarsenFactor;
  const float *B_lanes[CoarsenFactor];
#pragma unroll
  for (int i = 0; i < CoarsenFactor; i++) {
    B_lanes[i] = B + col_offset + lane_id + i * 32;
  }
  int ldB = N;

  // declare accumulators
  float c[CoarsenFactor] = {0.0f};
  int ldC = N;

  // N-dimension residual handling
  if (blockIdx.y == gridDim.y - 1)
    goto Ndim_Residue;

  // iterate over the sparse row
  for (int p = start; p < end; p += 32) {
    // copy a bucket of sparse row elements into shared memory
    if (p + lane_id < end) {
      workspace_data[lane_id] =
          __guard_load_default_one<float>(csr_data, (p + lane_id));
      workspace_indices[lane_id] = csr_indices[p + lane_id];
    } else {
      workspace_data[lane_id] = 0.0f;
      workspace_indices[lane_id] = 0;
    }
    __syncwarp();
// do MAC computation using buffered elements
#pragma unroll
    for (int pp = 0; pp < 32; pp++) {
      int k = workspace_indices[pp];
      float v = workspace_data[pp];
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        c[i] += v * B_lanes[i][k * ldB];
      }
    }
  }

// write results
#pragma unroll
  for (int i = 0; i < CoarsenFactor; i++) {
    float *C_lane = B_lanes[i] - B + (C + row_id * ldC);
    *C_lane = c[i];
  }
  return;

Ndim_Residue:
  int valid_lane_num = CEIL(N - col_offset - lane_id, 32);

  // iterate over the sparse row
  for (int p = start; p < end; p += 32) {
    // copy a bucket of sparse row elements into shared memory
    if (p + lane_id < end) {
      workspace_data[lane_id] =
          __guard_load_default_one<float>(csr_data, (p + lane_id));
      workspace_indices[lane_id] = csr_indices[p + lane_id];
    } else {
      workspace_data[lane_id] = 0.0f;
      workspace_indices[lane_id] = 0;
    }
    __syncwarp();
// do MAC computation using buffered elements
#pragma unroll
    for (int pp = 0; pp < 32; pp++) {
      int k = workspace_indices[pp];
      float v = workspace_data[pp];
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        if (i < valid_lane_num) {
          c[i] += v * B_lanes[i][k * ldB];
        }
      }
    }
  }

// write results
#pragma unroll
  for (int i = 0; i < CoarsenFactor; i++) {
    float *C_lane = B_lanes[i] - B + (C + row_id * ldC);
    if (i < valid_lane_num) {
      *C_lane = c[i];
    }
  }
  return;
}

template <int CoarsenFactor, int ThreadNz>
__global__ void csrspmm_rowcaching_nnzbalance_kernel(
    const int M, const int N, const int K, const int nnz_,
    const int csr_indptr[], const int csr_indices[], const float csr_data[],
    const float B[], float C[]) {
  int nnz = nnz_;
  if (nnz < 0)
    nnz = csr_indptr[M];

  int warp_id = threadIdx.x >> 5;
  int lane_id = threadIdx.x & 31;

  extern __shared__ int shared_mem[];
  int *workspace_rowid = &shared_mem[(warp_id << 5)];
  int *workspace_colid = workspace_rowid + blockDim.x;
  float *workspace_data =
      (float *)(workspace_colid +
                blockDim.x); // float and int has the same size

  // get the sparse-value range of this row
  int global_warp_id = blockIdx.x * (blockDim.x >> 5) + warp_id;
  int nz_start = global_warp_id * (ThreadNz * 32);

  // get the dense column offset
  int col_offset = blockIdx.y * 32 * CoarsenFactor;
  const float *B_lanes[CoarsenFactor];
  float *C_lanes[CoarsenFactor];
#pragma unroll
  for (int i = 0; i < CoarsenFactor; i++) {
    B_lanes[i] = B + col_offset + lane_id + i * 32;
    C_lanes[i] = C + col_offset + lane_id + i * 32;
  }
  int ldB = N;

  // declare accumulators
  float c[CoarsenFactor] = {0.0f};
  int ldC = N;

  int stride = gridDim.x * (blockDim.x >> 5) * ThreadNz * 32;

  if (blockIdx.y == gridDim.y - 1)
    goto Ndim_Residue;

  for (; nz_start < nnz; nz_start += stride) {
    // iterate over the segment of this warp
    for (int tile_base = nz_start;
         tile_base < min(nz_start + ThreadNz * 32, nnz); tile_base += 32) {
      int thread_nz_id = tile_base + lane_id;
      if (thread_nz_id < nnz) {
        workspace_colid[lane_id] = csr_indices[thread_nz_id];
        workspace_data[lane_id] =
            __guard_load_default_one<float>(csr_data, thread_nz_id);
      } else {
        workspace_colid[lane_id] = 0;
        workspace_data[lane_id] = 0.0f;
      }
      workspace_rowid[lane_id] =
          binary_search_segment_number<int>(csr_indptr, M, nnz, thread_nz_id);
      __syncwarp();

      // initialize with first value
      int k = workspace_colid[0];
      float v = workspace_data[0];
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        c[i] = v * B_lanes[i][k * ldB];
      }
      int row_curr = workspace_rowid[0], next_row;

// scan
#pragma unroll
      for (int pp = 1; pp < 32; pp++) {
        next_row = workspace_rowid[pp];
        if (next_row != row_curr) {
#pragma unroll
          for (int i = 0; i < CoarsenFactor; i++) {
            atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
          }
          row_curr = next_row;
          k = workspace_colid[pp];
          v = workspace_data[pp];
#pragma unroll
          for (int i = 0; i < CoarsenFactor; i++) {
            c[i] = v * B_lanes[i][k * ldB];
          }
        } else {
          k = workspace_colid[pp];
          v = workspace_data[pp];
#pragma unroll
          for (int i = 0; i < CoarsenFactor; i++) {
            c[i] = c[i] + v * B_lanes[i][k * ldB];
          }
        }
      }
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
      }
    }
  }
  return;

Ndim_Residue:

  int valid_lane_num = CEIL(N - col_offset - lane_id, 32);

  for (; nz_start < nnz; nz_start += stride) {
    // iterate over the segment of this warp
    for (int tile_base = nz_start;
         tile_base < min(nz_start + ThreadNz * 32, nnz); tile_base += 32) {
      int thread_nz_id = tile_base + lane_id;
      if (thread_nz_id < nnz) {
        workspace_colid[lane_id] = csr_indices[thread_nz_id];
        workspace_data[lane_id] =
            __guard_load_default_one<float>(csr_data, thread_nz_id);
      } else {
        workspace_colid[lane_id] = 0;
        workspace_data[lane_id] = 0.0f;
      }
      workspace_rowid[lane_id] =
          binary_search_segment_number<int>(csr_indptr, M, nnz, thread_nz_id);
      __syncwarp();

      // initialize with first value
      int k = workspace_colid[0];
      float v = workspace_data[0];
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        if (i < valid_lane_num) {
          c[i] = v * B_lanes[i][k * ldB];
        }
      }
      int row_curr = workspace_rowid[0], next_row;

// scan
#pragma unroll
      for (int pp = 1; pp < 32; pp++) {
        next_row = workspace_rowid[pp];
        if (next_row != row_curr) {
#pragma unroll
          for (int i = 0; i < CoarsenFactor; i++) {
            if (i < valid_lane_num) {
              atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
            }
          }
          row_curr = next_row;
          k = workspace_colid[pp];
          v = workspace_data[pp];
#pragma unroll
          for (int i = 0; i < CoarsenFactor; i++) {
            if (i < valid_lane_num) {
              c[i] = v * B_lanes[i][k * ldB];
            }
          }
        } else {
          k = workspace_colid[pp];
          v = workspace_data[pp];
#pragma unroll
          for (int i = 0; i < CoarsenFactor; i++) {
            if (i < valid_lane_num) {
              c[i] = c[i] + v * B_lanes[i][k * ldB];
            }
          }
        }
      }
#pragma unroll
      for (int i = 0; i < CoarsenFactor; i++) {
        if (i < valid_lane_num) {
          atomicAdd(C_lanes[i] + row_curr * ldC, c[i]);
        }
      }
    }
  }
}

void csrspmm_rowcaching_rowbalance(const SpMatCsrDescr_t spmatA, const float *B,
                                   const int N, float *C) {
  int coarsen_factor = (N >= 512) ? 4 : (N >= 128) ? 2 : 1;
  int Ndim_threadblock = CEIL(N, (32 * coarsen_factor));
  int Mdim_warp_per_tb = RefThreadPerBlock / 32;
  dim3 gridDim(CEIL(spmatA.nrow, Mdim_warp_per_tb), Ndim_threadblock, 1);
  dim3 blockDim(RefThreadPerBlock, 1, 1);

  size_t smem_size = (sizeof(int) + sizeof(float)) * RefThreadPerBlock;

  if (coarsen_factor == 4) {
    csrspmm_rowcaching_rowbalance_kernel<4><<<gridDim, blockDim, smem_size>>>(
        spmatA.nrow, N, spmatA.ncol, spmatA.indptr, spmatA.indices, spmatA.data,
        B, C);
  } else if (coarsen_factor == 2) {
    csrspmm_rowcaching_rowbalance_kernel<2><<<gridDim, blockDim, smem_size>>>(
        spmatA.nrow, N, spmatA.ncol, spmatA.indptr, spmatA.indices, spmatA.data,
        B, C);
  } else {
    csrspmm_rowcaching_rowbalance_kernel<1><<<gridDim, blockDim, smem_size>>>(
        spmatA.nrow, N, spmatA.ncol, spmatA.indptr, spmatA.indices, spmatA.data,
        B, C);
  }
}

void csrspmm_rowcaching_nnzbalance(const SpMatCsrDescr_t spmatA, const float *B,
                                   const int N, float *C) {
  int coarsen_factor = (N >= 512) ? 4 : (N >= 128) ? 2 : 1;
  int Ndim_threadblock = CEIL(N, (32 * coarsen_factor));

  // int thread_nz = (spmatA.nnz > 8000 * 128 * 2) ? 2 : 1;
  int thread_nz = 1;
  int Nnzdim_warp_per_tb = RefThreadPerBlock / 32;
  // int Nnzdim_threadblock = CEIL(spmatA.nnz, Nnzdim_warp_per_tb * 32 *
  // thread_nz );
  int Nnzdim_threadblock = CEIL(
      spmatA.nrow,
      Nnzdim_warp_per_tb * thread_nz); // CEIL(spmatA.nnz, Nnzdim_warp_per_tb *
                                       // 32 * thread_nz );

  dim3 gridDim(Nnzdim_threadblock, Ndim_threadblock, 1);
  dim3 blockDim(RefThreadPerBlock, 1, 1);

  size_t smem_size = (2 * sizeof(int) + sizeof(float)) * RefThreadPerBlock;

  // simple heuristic

  if (coarsen_factor == 4) {
    if (thread_nz == 1)
      csrspmm_rowcaching_nnzbalance_kernel<4, 1>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
    if (thread_nz == 2)
      csrspmm_rowcaching_nnzbalance_kernel<4, 2>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
    if (thread_nz == 4)
      csrspmm_rowcaching_nnzbalance_kernel<4, 4>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
  } else if (coarsen_factor == 2) {
    if (thread_nz == 1)
      csrspmm_rowcaching_nnzbalance_kernel<2, 1>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
    if (thread_nz == 2)
      csrspmm_rowcaching_nnzbalance_kernel<2, 2>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
    if (thread_nz == 4)
      csrspmm_rowcaching_nnzbalance_kernel<2, 4>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
  } else {
    if (thread_nz == 1)
      csrspmm_rowcaching_nnzbalance_kernel<1, 1>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
    if (thread_nz == 2)
      csrspmm_rowcaching_nnzbalance_kernel<1, 2>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
    if (thread_nz == 4)
      csrspmm_rowcaching_nnzbalance_kernel<1, 4>
          <<<gridDim, blockDim, smem_size>>>(spmatA.nrow, N, spmatA.ncol,
                                             spmatA.nnz, spmatA.indptr,
                                             spmatA.indices, spmatA.data, B, C);
  }
}
