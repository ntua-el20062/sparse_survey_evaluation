#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "macros/cpp_defines.h"

#include "bench_common.h"
#include "kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"
	#include "array_metrics.h"

	#include "cuda/cuda_util.h"
	#include "cuda/cusparse_util.h"
#ifdef __cplusplus
}
#endif

#if DOUBLE == 0
	#define ValueTypeCuda  HIP_R_32F
#elif DOUBLE == 1
	#define ValueTypeCuda  HIP_R_64F
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;

	hipStream_t stream;
	hipsparseHandle_t     handle = NULL;
	hipsparseSpMatDescr_t matA;
	void*                dBuffer    = NULL;
	size_t               bufferSize = 0;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * out = NULL;
	
	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	
	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	ValueType * out_h = NULL;
	
	hipsparseDnMatDescr_t matX;
	hipsparseDnMatDescr_t matY;

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipMalloc((void**)&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&a_d, nnz * sizeof(*a_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));
		gpuCusparseErrorCheck(hipsparseCreate(&handle));
		gpuCusparseErrorCheck(hipsparseSetStream(handle, stream));

		gpuCudaErrorCheck(hipHostMalloc((void**)&ia_h, (m+1) * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&ja_h, nnz * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&a_h, nnz * sizeof(*a_h)));

		memcpy(ia_h, ia, (m+1) * sizeof(*ia_h));
		memcpy(ja_h, ja, nnz * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));

		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));

		// wait for transfers to finish
		gpuCudaErrorCheck(hipStreamSynchronize(stream));

		// Create sparse matrix A in CSR format
		gpuCusparseErrorCheck(hipsparseCreateCsr(&matA, m, n, nnz, ia_d, ja_d, a_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, ValueTypeCuda));
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));
		gpuCudaErrorCheck(hipHostFree(out_h));

		gpuCusparseErrorCheck(hipsparseDestroySpMat(matA));
		gpuCusparseErrorCheck(hipsparseDestroyDnMat(matX));
		gpuCusparseErrorCheck(hipsparseDestroyDnMat(matY));

		gpuCusparseErrorCheck(hipsparseDestroy(handle));
		gpuCudaErrorCheck(hipStreamDestroy(stream));
		gpuCudaErrorCheck(hipFree(dBuffer));
	}

	void spmm(ValueType * x, ValueType * y, int k);
	void sddmm(ValueType * x, ValueType * y, ValueType * out, int k);
};

void compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k);
void compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k);

void
CSRArrays::spmm(ValueType * x, ValueType * y, int k)
{
	compute_spmm(this, x, y, k);
}

void
CSRArrays::sddmm(ValueType * x, ValueType * y, ValueType * out, int k)
{
	compute_sddmm(this, x, y, out, k);
}

struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "CUSPARSE_CSR";
	return csr;
}

//==========================================================================================================================================
//= Computation
//==========================================================================================================================================

void
compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k)
{
	const ValueType alpha = 1.0;
	const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;

		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->n * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->n * k * sizeof(*csr->x_h)));

		memcpy(csr->x_h, x, csr->n * k * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Create dense matrix X
		gpuCusparseErrorCheck(hipsparseCreateDnMat(&csr->matX, csr->n, k, k, csr->x_d, ValueTypeCuda, HIPSPARSE_ORDER_ROW)); // HIPSPARSE_ORDER_COL

		// Also, prepare for the output matrix y
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, csr->m * k * sizeof(*csr->y_d)));

		// Create dense matrix Y
		gpuCusparseErrorCheck(hipsparseCreateDnMat(&csr->matY, csr->m, k, k, csr->y_d, ValueTypeCuda, HIPSPARSE_ORDER_ROW)); // HIPSPARSE_ORDER_COL

		// Allocate an external buffer if needed and finish preprocessing
		gpuCusparseErrorCheck(hipsparseSpMM_bufferSize(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->matX, &beta, csr->matY, ValueTypeCuda, HIPSPARSE_SPMM_ALG_DEFAULT, &csr->bufferSize));
		gpuCudaErrorCheck(hipMalloc((void**)&csr->dBuffer, csr->bufferSize));

		gpuCusparseErrorCheck(hipsparseSpMM_preprocess(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->matX, &beta, csr->matY, ValueTypeCuda, HIPSPARSE_SPMM_ALG_DEFAULT, csr->dBuffer));
	}

	gpuCusparseErrorCheck(hipsparseSpMM(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matA, csr->matX, &beta, csr->matY, ValueTypeCuda, HIPSPARSE_SPMM_ALG_DEFAULT, csr->dBuffer));

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, csr->m * k * sizeof(*csr->y_h)));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * k * sizeof(*csr->y_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * k * sizeof(ValueType));
	}
}

void
compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k)
{
	const ValueType alpha = 1.0;
	const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		csr->y = y;

		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->m * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, k * csr->n * sizeof(*csr->y_d)));

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->m * k * sizeof(*csr->x_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, k * csr->n * sizeof(*csr->y_h)));

		memcpy(csr->x_h, x, csr->m * k * sizeof(ValueType));
		memcpy(csr->y_h, y, k * csr->n * sizeof(ValueType));

		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->m * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_d, csr->y_h, k * csr->n * sizeof(*csr->y_d), hipMemcpyHostToDevice, csr->stream));

		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Create dense matrices X and Y
		gpuCusparseErrorCheck(hipsparseCreateDnMat(&csr->matX, csr->m, k, k, csr->x_d, ValueTypeCuda, HIPSPARSE_ORDER_ROW)); // HIPSPARSE_ORDER_COL
		gpuCusparseErrorCheck(hipsparseCreateDnMat(&csr->matY, k, csr->n, csr->n, csr->y_d, ValueTypeCuda, HIPSPARSE_ORDER_ROW)); // HIPSPARSE_ORDER_COL

		// Allocate an external buffer if needed and finish preprocessing
		gpuCusparseErrorCheck(hipsparseSDDMM_bufferSize(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matX, csr->matY, &beta, csr->matA, ValueTypeCuda, HIPSPARSE_SDDMM_ALG_DEFAULT, &csr->bufferSize))
		gpuCudaErrorCheck(hipMalloc((void**)&csr->dBuffer, csr->bufferSize));

		gpuCusparseErrorCheck(hipsparseSDDMM_preprocess(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matX, csr->matY, &beta, csr->matA, ValueTypeCuda, HIPSPARSE_SDDMM_ALG_DEFAULT, csr->dBuffer))
	}

	gpuCusparseErrorCheck(hipsparseSDDMM(csr->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csr->matX, csr->matY, &beta, csr->matA, ValueTypeCuda, HIPSPARSE_SDDMM_ALG_DEFAULT, csr->dBuffer))

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->out == NULL)
	{
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->out_h, csr->nnz * sizeof(*csr->out_h)));

		csr->out = out;

		gpuCudaErrorCheck(hipMemcpyAsync(csr->out_h, csr->a_d, csr->nnz * sizeof(*csr->a_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(out, csr->out_h, csr->nnz * sizeof(ValueType));
	}
}
