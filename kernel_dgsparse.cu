#include <stdio.h>
#include <stdlib.h>

#include "macros/cpp_defines.h"

#include "bench_common.h"
#include "kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

#include <hip/hip_runtime.h>

#ifdef SPMM_KERNEL
	#include "gespmm.h"
#endif
#ifdef SDDMM_KERNEL
	#include "sddmm.h"
#endif

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;

	hipStream_t stream;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * out = NULL;

	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	// ValueType * out_d = NULL;

	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	ValueType * out_h = NULL;

	#ifdef SPMM_KERNEL
		SpMatCsrDescr_t spmatA;
		gespmmAlg_t method;

		// this will be the fallback method (the first one), in case it is not specified
		#ifndef SPMM_METHOD
			#define SPMM_METHOD 0
		#endif
	#endif

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipMalloc((void**)&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&a_d, nnz * sizeof(*a_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));

		gpuCudaErrorCheck(hipHostMalloc((void**)&ia_h, (m+1) * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&ja_h, nnz * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&a_h, nnz * sizeof(*a_h)));

		memcpy(ia_h, ia, (m+1) * sizeof(*ia_h));
		memcpy(ja_h, ja, nnz * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));

		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));

		// wait for transfers to finish
		gpuCudaErrorCheck(hipStreamSynchronize(stream));

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		
		// Preprocessing takes place only for the SpMM kernel, only its header file contains the necessary struct definitions.		
		#ifdef SPMM_KERNEL
		spmatA = SpMatCsrDescr_t{(int) m, (int) n, (int) nnz, ia_d, ja_d, a_d};
		gespmmAlg_t algs[] = {
				GESPMM_ALG_SEQREDUCE_ROWBALANCE,  GESPMM_ALG_PARREDUCE_ROWBALANCE,
				GESPMM_ALG_SEQREDUCE_NNZBALANCE,  GESPMM_ALG_PARREDUCE_NNZBALANCE,
				GESPMM_ALG_ROWCACHING_ROWBALANCE, GESPMM_ALG_ROWCACHING_NNZBALANCE
			};

		int method_int = SPMM_METHOD;
		method = algs[method_int];
		#endif

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		// gpuCudaErrorCheck(hipFree(out_d));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));
		gpuCudaErrorCheck(hipHostFree(out_h));

		gpuCudaErrorCheck(hipStreamDestroy(stream));
	}

	void spmm(ValueType * x, ValueType * y, int k);
	void sddmm(ValueType * x, ValueType * y, ValueType * out, int k);
};

void compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k);
void compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k);

void
CSRArrays::spmm(ValueType * x, ValueType * y, int k)
{
	compute_spmm(this, x, y, k);
}

void
CSRArrays::sddmm(ValueType * x, ValueType * y, ValueType * out, int k)
{
	compute_sddmm(this, x, y, out, k);
}

struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	#ifdef SPMM_KERNEL
	char buffer[64];
	snprintf(buffer, sizeof(buffer), "dgSPARSE-%d", SPMM_METHOD);
	csr->format_name = strdup(buffer);
	#else
	csr->format_name = (char *) "dgSPARSE";
	#endif
	return csr;
}

//==========================================================================================================================================
//= Computation
//==========================================================================================================================================

void
compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k)
{
	__attribute__((unused)) const ValueType alpha = 1.0;
	__attribute__((unused)) const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->n * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->n * k * sizeof(*csr->x_h)));

		memcpy(csr->x_h, x, csr->n * k * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Also, prepare for the output matrix y
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, csr->m * k * sizeof(*csr->y_d)));
	}

	#ifdef SPMM_KERNEL
	gespmmCsrSpMM(csr->spmatA, csr->x_d, k, csr->y_d, true, csr->method);
	#endif

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, csr->m * k * sizeof(*csr->y_h)));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * k * sizeof(*csr->y_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * k * sizeof(ValueType));
	}
}

void
compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k)
{
	__attribute__((unused)) const ValueType alpha = 1.0;
	__attribute__((unused)) const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		csr->y = y;

		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->m * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, k * csr->n * sizeof(*csr->y_d)));

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->m * k * sizeof(*csr->x_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, k * csr->n * sizeof(*csr->y_h)));

		memcpy(csr->x_h, x, csr->m * k * sizeof(ValueType));
		memcpy(csr->y_h, y, k * csr->n * sizeof(ValueType));

		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->m * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_d, csr->y_h, k * csr->n * sizeof(*csr->y_d), hipMemcpyHostToDevice, csr->stream));

		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
	}

	#ifdef SDDMM_KERNEL
	sddmm_cuda_csr(csr->m, k, csr->nnz, csr->ia_d, csr->ja_d, csr->x_d, csr->y_d, csr->a_d);
	#endif

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->out == NULL)
	{
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->out_h, csr->nnz * sizeof(*csr->out_h)));

		csr->out = out;

		gpuCudaErrorCheck(hipMemcpyAsync(csr->out_h, csr->a_d, csr->nnz * sizeof(*csr->a_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(out, csr->out_h, csr->nnz * sizeof(ValueType));
	}
}
