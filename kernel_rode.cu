#include <stdio.h>
#include <stdlib.h>

#include "macros/cpp_defines.h"

#include "bench_common.h"
#include "kernel.h"

#ifdef __cplusplus
extern "C"{
#endif
	#include "macros/macrolib.h"
	#include "time_it.h"
	#include "parallel_util.h"

	#include "cuda/cuda_util.h"
#ifdef __cplusplus
}
#endif

#include <hip/hip_runtime.h>

#include <memory>
#ifdef SPMM_KERNEL
	#include "spmm/RoDeSpmm.h"
	#include "spmm/matrix_utils.h"
	// using namespace SPC;
#endif
#ifdef SDDMM_KERNEL
	#include "sddmm/RoDeSddmm.h"
	#include "sddmm/matrix_utils.h"
	// using namespace SPC;
#endif
using namespace SPC;

struct CSRArrays : Matrix_Format
{
	INT_T * ia;      // the usual rowptr (of size m+1)
	INT_T * ja;      // the colidx of each NNZ (of size nnz)
	ValueType * a;   // the values (of size NNZ)

	INT_T * ia_d;
	INT_T * ja_d;
	ValueType * a_d;

	INT_T * ia_h;
	INT_T * ja_h;
	ValueType * a_h;

	hipStream_t stream;

	ValueType * x = NULL;
	ValueType * y = NULL;
	ValueType * out = NULL;

	ValueType * x_d = NULL;
	ValueType * y_d = NULL;
	ValueType * out_d = NULL;

	ValueType * x_h = NULL;
	ValueType * y_h = NULL;
	ValueType * out_h = NULL;

	hipStream_t stream1, stream2;

	#if defined(SPMM_KERNEL) || defined(SDDMM_KERNEL)
		std::unique_ptr<SPC::SparseMatrix> sm1;
		std::unique_ptr<SPC::CudaSparseMatrix<ValueType>> c_sm;
	#endif

	CSRArrays(INT_T * ia, INT_T * ja, ValueType * a, long m, long n, long nnz) : Matrix_Format(m, n, nnz), ia(ia), ja(ja), a(a)
	{
		gpuCudaErrorCheck(hipMalloc((void**)&ia_d, (m+1) * sizeof(*ia_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&ja_d, nnz * sizeof(*ja_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&a_d, nnz * sizeof(*a_d)));

		gpuCudaErrorCheck(hipStreamCreate(&stream));

		gpuCudaErrorCheck(hipHostMalloc((void**)&ia_h, (m+1) * sizeof(*ia_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&ja_h, nnz * sizeof(*ja_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&a_h, nnz * sizeof(*a_h)));

		memcpy(ia_h, ia, (m+1) * sizeof(*ia_h));
		memcpy(ja_h, ja, nnz * sizeof(*ja_h));
		memcpy(a_h, a, nnz * sizeof(*a_h));

		gpuCudaErrorCheck(hipMemcpyAsync(ia_d, ia_h, (m+1) * sizeof(*ia_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(ja_d, ja_h, nnz * sizeof(*ja_d), hipMemcpyHostToDevice, stream));
		gpuCudaErrorCheck(hipMemcpyAsync(a_d, a_h, nnz * sizeof(*a_d), hipMemcpyHostToDevice, stream));

		// wait for transfers to finish
		gpuCudaErrorCheck(hipStreamSynchronize(stream));

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		gpuCudaErrorCheck(hipStreamCreate(&stream1));
		gpuCudaErrorCheck(hipStreamCreate(&stream2));

		// The sparse matrix will be read here, that's why no previous proper reading in row_ptr, col_idx, val... leaving it for legacy reasons

		#if defined(SPMM_KERNEL) || defined(SDDMM_KERNEL)
			sm1 = std::make_unique<SPC::SparseMatrix>(ia, ja, a, m, n, nnz, SPC::SORTED, 1);
			sm1->RowDivide2Segment(512,4,32); // why this 512? nobody will know...
			c_sm = std::make_unique<SPC::CudaSparseMatrix<ValueType>>(*sm1);
		#endif

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	}

	~CSRArrays()
	{
		free(a);
		free(ia);
		free(ja);

		gpuCudaErrorCheck(hipFree(ia_d));
		gpuCudaErrorCheck(hipFree(ja_d));
		gpuCudaErrorCheck(hipFree(a_d));
		gpuCudaErrorCheck(hipFree(x_d));
		gpuCudaErrorCheck(hipFree(y_d));
		gpuCudaErrorCheck(hipFree(out_d));

		gpuCudaErrorCheck(hipHostFree(ia_h));
		gpuCudaErrorCheck(hipHostFree(ja_h));
		gpuCudaErrorCheck(hipHostFree(a_h));
		gpuCudaErrorCheck(hipHostFree(x_h));
		gpuCudaErrorCheck(hipHostFree(y_h));
		gpuCudaErrorCheck(hipHostFree(out_h));

		gpuCudaErrorCheck(hipStreamDestroy(stream));
		
		gpuCudaErrorCheck(hipStreamDestroy(stream1));
		gpuCudaErrorCheck(hipStreamDestroy(stream2));
	}

	void spmm(ValueType * x, ValueType * y, int k);
	void sddmm(ValueType * x, ValueType * y, ValueType * out, int k);
};

void compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k);
void compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k);

void
CSRArrays::spmm(ValueType * x, ValueType * y, int k)
{
	compute_spmm(this, x, y, k);
}

void
CSRArrays::sddmm(ValueType * x, ValueType * y, ValueType * out, int k)
{
	compute_sddmm(this, x, y, out, k);
}

struct Matrix_Format *
csr_to_format(INT_T * row_ptr, INT_T * col_ind, ValueType * values, long m, long n, long nnz)
{
	struct CSRArrays * csr = new CSRArrays(row_ptr, col_ind, values, m, n, nnz);
	csr->mem_footprint = nnz * (sizeof(ValueType) + sizeof(INT_T)) + (m+1) * sizeof(INT_T);
	csr->format_name = (char *) "RoDe";
	return csr;
}

//==========================================================================================================================================
//= Computation
//==========================================================================================================================================

void
compute_spmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, int k)
{
	__attribute__((unused)) const ValueType alpha = 1.0;
	__attribute__((unused)) const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->n * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->n * k * sizeof(*csr->x_h)));

		memcpy(csr->x_h, x, csr->n * k * sizeof(ValueType));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->n * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Also, prepare for the output matrix y
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, csr->m * k * sizeof(*csr->y_d)));
	}

	#ifdef SPMM_KERNEL
	// Perhaps 128 is faster? Don't know for sure...
	// RoDeSpmm_n32(csr->c_sm->n_segs, csr->c_sm->n_segs_residue, csr->n, k, csr->c_sm->Values(), csr->c_sm->ColumnIndices(), csr->c_sm->RowOffsets(), csr->c_sm->seg_row_indices, csr->c_sm->seg_row_indices_residue, csr->c_sm->seg_st_offsets, csr->x_d, csr->y_d, csr->stream1, csr->stream2);
	RoDeSpmm_n128(csr->c_sm->n_segs, csr->c_sm->n_segs_residue, csr->n, k, csr->c_sm->Values(), csr->c_sm->ColumnIndices(), csr->c_sm->RowOffsets(), csr->c_sm->seg_row_indices, csr->c_sm->seg_row_indices_residue, csr->c_sm->seg_st_offsets, csr->x_d, csr->y_d, csr->stream1, csr->stream2);
	#endif

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->y == NULL)
	{
		csr->y = y;

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, csr->m * k * sizeof(*csr->y_h)));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_h, csr->y_d, csr->m * k * sizeof(*csr->y_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(y, csr->y_h, csr->m * k * sizeof(ValueType));
	}
}

void
compute_sddmm(CSRArrays * restrict csr, ValueType * restrict x, ValueType * restrict y, ValueType * restrict out, int k)
{
	__attribute__((unused)) const ValueType alpha = 1.0;
	__attribute__((unused)) const ValueType beta = 0.0;
	if (csr->x == NULL)
	{
		csr->x = x;
		csr->y = y;

		gpuCudaErrorCheck(hipMalloc((void**)&csr->x_d, csr->m * k * sizeof(*csr->x_d)));
		gpuCudaErrorCheck(hipMalloc((void**)&csr->y_d, k * csr->n * sizeof(*csr->y_d)));

		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->x_h, csr->m * k * sizeof(*csr->x_h)));
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->y_h, k * csr->n * sizeof(*csr->y_h)));

		memcpy(csr->x_h, x, csr->m * k * sizeof(ValueType));
		memcpy(csr->y_h, y, k * csr->n * sizeof(ValueType));

		gpuCudaErrorCheck(hipMemcpyAsync(csr->x_d, csr->x_h, csr->m * k * sizeof(*csr->x_d), hipMemcpyHostToDevice, csr->stream));
		gpuCudaErrorCheck(hipMemcpyAsync(csr->y_d, csr->y_h, k * csr->n * sizeof(*csr->y_d), hipMemcpyHostToDevice, csr->stream));

		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));

		// Also, prepare for the output values
		gpuCudaErrorCheck(hipMalloc((void**)&csr->out_d, csr->nnz * sizeof(*csr->out_d)));
	}

	#ifdef SDDMM_KERNEL
	// Each version is producing wrong results... 32 always->32, 128 always 128... Whatever, moving on...
	RoDeSDDMM_n32(csr->c_sm->n_segs, csr->c_sm->n_segs_residue, csr->n, k, csr->c_sm->seg_row_indices, csr->c_sm->seg_row_indices_residue, csr->c_sm->seg_st_offsets, csr->c_sm->RowOffsets(), csr->c_sm->ColumnIndices(), csr->c_sm->Values(), csr->x_d, csr->y_d, csr->out_d, csr->stream1, csr->stream2);
	// RoDeSDDMM_n128(csr->c_sm->n_segs, csr->c_sm->n_segs_residue, csr->n, k, csr->c_sm->seg_row_indices, csr->c_sm->seg_row_indices_residue, csr->c_sm->seg_st_offsets, csr->c_sm->RowOffsets(), csr->c_sm->ColumnIndices(), csr->c_sm->Values(), csr->x_d, csr->y_d, csr->out_d, csr->stream1, csr->stream2);
	#endif

	gpuCudaErrorCheck(hipPeekAtLastError());
	gpuCudaErrorCheck(hipDeviceSynchronize());

	if (csr->out == NULL)
	{
		gpuCudaErrorCheck(hipHostMalloc((void**)&csr->out_h, csr->nnz * sizeof(*csr->out_h)));

		csr->out = out;

		gpuCudaErrorCheck(hipMemcpyAsync(csr->out_h, csr->out_d, csr->nnz * sizeof(*csr->out_d), hipMemcpyDeviceToHost, csr->stream));
		gpuCudaErrorCheck(hipStreamSynchronize(csr->stream));
		memcpy(out, csr->out_h, csr->nnz * sizeof(ValueType));
	}
}
